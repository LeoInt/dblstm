#include "hip/hip_runtime.h"
#include <float.h>
#include <math.h>
#include <iostream>

#include "kernels.h"
// Device functions
__forceinline__ __device__ float sigmoidf(float in) {
  return 1.f / (1.f + expf(-in));  
}



__global__ void rand_generate(fp16 *y,  int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = fp32_to_fp16_gpu(sinf(i)/10);
}

__global__ void pw_biasAdd(float *y, float *bias, int n, int nBias) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] += bias[i % nBias];
}

__global__ void pw_vecAdd(float *y, float *a,  float *b, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = a[i] + b[i];
}

__global__ void pw_vecMul(float *y, float *a,  float *b, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = a[i] * b[i];
}

__global__ void pw_tanh(float *y, float *a, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = tanh(a[i]);
}

__global__ void pw_sigmoid(float *y, float *a, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = sigmoidf(a[i]);
}
__global__ void add_pw_vecMul(float *y, float *a,  float *b, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = y[i] + a[i] * b[i];
}

__global__ void initKernel(float * devPtr, const int val, int nwords){
  int tidx = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;

  for(; tidx < nwords; tidx += stride)
    devPtr[tidx] = val;
}


__global__ void testfloat(float * devPtr, float * devPtr1){
  int tidx = threadIdx.x + blockDim.x * blockIdx.x;
  devPtr[0]=devPtr[0]*devPtr1[0];
}

__global__ void elementWise_fp(int hiddenSize, float *tmp_h, float *tmp_i, float *bias, float *phole_i, float *phole_f, float *phole_o, float *h_out, float *i_out, float *c_in, float *c_out) {  
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   
   if (index >= hiddenSize) return;
   
   //int batch = index / hiddenSize;
   int gateIndex = (index % hiddenSize); //+ 4 * batch * hiddenSize;   
   
   float g[4];

   for (int i = 0; i < 4; i++) {
      g[i] = tmp_i[i * hiddenSize + gateIndex] + tmp_h[i * hiddenSize + gateIndex];
      g[i] += bias[i * hiddenSize + index % hiddenSize];
   }  

    g[1] += c_in[index]*phole_i[index];
    g[2] += c_in[index]*phole_f[index];
    //g[3] += c_in[index]*phole_o[index];
        
   
   float in_gate2    = tanhf(g[0]);   
   float in_gate     = sigmoidf(g[1]);
   float forget_gate = sigmoidf(g[2]);

   float out_gate    = g[3];
   
   float val = (forget_gate * c_in[index]) + (in_gate * in_gate2);
     
   c_out[index] = val;
   out_gate += val*phole_o[index];
   out_gate = sigmoidf(out_gate);
   val = out_gate * tanhf(val);                                   

   h_out[index] = val;
   i_out[index] = val;
}


// Fused forward kernel
__global__ void elementWise_fp_1(int hiddenSize,
                               float *tmp_h, 
                               float *tmp_i, 
                               float *bias,
                               float *phole_i,
                               float *phole_f,
                               float *c_in) {
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   
   if (index >= hiddenSize*4) return;
   
   int gateIndex = (index % hiddenSize);// + 4 * batch * hiddenSize;   
   
   float g;
   g = tmp_i[index] + tmp_h[index] + bias[index];
   
   if(index < hiddenSize ){
     //g += c_in[gateIndex]*phole_i[gateIndex];
     tmp_i[index]=tanhf(g); //ingate2
   } else if(index >=hiddenSize && index <2*hiddenSize ){
    g += c_in[gateIndex]*phole_i[gateIndex]; 
    tmp_i[index]=sigmoidf(g); //input
   }else if(index >=2*hiddenSize && index <3*hiddenSize ){
    g += c_in[gateIndex]*phole_f[gateIndex];
    tmp_i[index]=sigmoidf(g); //forget
   }else{
    //g += c_in[gateIndex]*phole_o[gateIndex];
    tmp_i[index]=g; //out
   }


}   
 __global__ void elementWise_fp_2(int hiddenSize,
                               float *tmp_i, 
                               float *c_in,
                               float *phole_o,
                               float *h_out,
                               float *i_out,
                               float *c_out) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
   float out_gate;
   if (index >= hiddenSize) return;
   
   float val = (tmp_i[2*hiddenSize + index] * c_in[index]) + (tmp_i[index] * tmp_i[index+hiddenSize]);
   
   c_out[index] = val;
   out_gate =tmp_i[index+3*hiddenSize]; 
   out_gate += val*phole_o[index];
   val = sigmoidf(out_gate) * tanhf(val);                                   

   h_out[index] = val;
   i_out[index] = val;
}

__global__ void add_vec_to_rows(float alpha, float* row, float beta, float* dst, int nrow, int ncol) { //46(outdim) col and 100 row(seqlength) in row major, 46 rrow and 100 col in col major 
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int index = i + j*ncol;
  if (i < ncol && j < nrow)
    dst[index] = alpha*row[i] + beta*dst[index];
}
//y = e^x_j/sum_j(e^x_j)
__global__ void _softmax_reduce(float*y, float*x, int nrow, int ncol) { //nrow = seqLength, ncol=output_dim.
  int j = blockIdx.x;
  int THREADS = blockDim.x;
  if (j >= nrow) return;

  __shared__ float aux[CU1DBLOCK];
  int steps = (ncol - 1) / THREADS + 1;

  //copy input to aux
  aux[threadIdx.x] = x[threadIdx.x+j*ncol]; //stride=ncol
  for(int i=1; i<steps; ++i) {
    if(threadIdx.x+i*THREADS < ncol && aux[threadIdx.x] < x[threadIdx.x+i*THREADS+j*ncol])
  aux[threadIdx.x] = x[threadIdx.x+i*THREADS+j*ncol];
  }

  //get the maximum value
  int nTotalThreads = THREADS;
  __syncthreads();
  while(nTotalThreads > 1) {
    int halfPoint = ((1+nTotalThreads) >> 1);   // divide by two
    // only the first half of the threads will be active.
    if (threadIdx.x < halfPoint)  {
      // Get the shared value stored by another thread
      if(threadIdx.x+halfPoint < nTotalThreads && aux[threadIdx.x] < aux[threadIdx.x+halfPoint])
        aux[threadIdx.x] = aux[threadIdx.x + halfPoint];
    }
    __syncthreads();
    nTotalThreads = ((1+nTotalThreads) >> 1);   // divide by two.
  }
  float max = aux[0];
  __syncthreads();
  
   // subtract max, apply exp, sum up...
  y[threadIdx.x+j*ncol] = exp(x[threadIdx.x+j*ncol] - max);
  aux[threadIdx.x] = y[threadIdx.x+j*ncol];
  for(int i=1; i<steps; i++) {
    if(threadIdx.x+i*THREADS < ncol) {
      y[threadIdx.x+i*THREADS+j*ncol] = exp(x[threadIdx.x+i*THREADS+j*ncol] - max);
      aux[threadIdx.x] += y[threadIdx.x+i*THREADS+j*ncol];
    }
  }
  nTotalThreads = THREADS;
  __syncthreads();
  while(nTotalThreads > 1) {
    int halfPoint = ((1+nTotalThreads) >> 1);   // divide by two
    // only the first half of the threads will be active.
    if (threadIdx.x < halfPoint)  {
      // Get the shared value stored by another thread
      if(threadIdx.x+halfPoint < nTotalThreads)
        aux[threadIdx.x] += aux[threadIdx.x + halfPoint];
    }
    __syncthreads();
    nTotalThreads = ((1+nTotalThreads) >> 1);   // divide by two.
  }
  float sum = aux[0];
  __syncthreads();

  //normalize by sum...
  for(int i=0; i<steps; i++) {
    if(threadIdx.x+i*THREADS < ncol) {
      y[threadIdx.x+i*THREADS+j*ncol] = y[threadIdx.x+i*THREADS+j*ncol] / sum;
    }
  }

}

__global__ void matvec_16(const fp16*   dA, const float*  dx,
    float*  dy, const uint_t nRows, const uint_t nx)
{
  const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  __shared__ float x_shared[BLOCK_SIZE];

  float y_val = 0.0;
//  float y_val2 = 0.0;
  
  //y_val=fp16_to_fp32_gpu(fp32_to_fp16_gpu(y_val));
  #pragma unroll
  for (unsigned int m = 0; m < ((nx + BLOCK_SIZE - 1) / BLOCK_SIZE); ++m) {

    if ((m * BLOCK_SIZE + threadIdx.x) < nx)
      x_shared[threadIdx.x] = dx[threadIdx.x + m * BLOCK_SIZE];
    else
      x_shared[threadIdx.x] = 0.f;

    __syncthreads();

    #pragma unroll
    for (unsigned int e = 0; e < BLOCK_SIZE; ++e) {
      float t = fp16_to_fp32_gpu(dA[tid + (e + BLOCK_SIZE * m) * nRows]);
      y_val += t * x_shared[e];
    }

    __syncthreads();
  }

  if (tid < nRows){
    dy[tid] = y_val;
  }
} /* End function matvec_kernel */

__global__ void matmat_16(const fp16*  __restrict__ A, const float*  __restrict__ B,
    float*  __restrict__ C, const uint_t M, const uint_t K, const uint_t N)
{
    //#define SUB_M  64//subultiple pf M  //rule : SUB_M=SUB_K and SUB_N * SUB_K < 1024
    //#define SUB_N  16//submultiple of N
    //#define SUB_K  64//submultple of K

  const int tid = threadIdx.x; 
  const unsigned int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  const uint_t K_sub_blocks = (K + SUB_K - 1) / SUB_K;
  float out[SUB_N];

  __shared__ float B_shared[SUB_N][SUB_K];
  
  #pragma unroll  
  for(int i=0; i<SUB_N; i++){
    out[i]=0;
  }

  register float y_val = 0.0;

  //#pragma unroll
  for (unsigned int m = 0; m < K_sub_blocks; ++m) {
    //#pragma unroll
    for(int a=0;a<SUB_N;a++){
        if(blockIdx.y*SUB_N+a<N)
            B_shared[a][tid]=B[blockIdx.y*K*SUB_N +a*K + m*SUB_K+tid];
        else
            B_shared[a][tid]=0;
    }
    __syncthreads();

    //#pragma unroll
    for (unsigned int e = 0; e < SUB_K; ++e) {
      //if((SUB_K * m + e)<K){
        float temp = fp16_to_fp32_gpu(A[tidx + (SUB_K * m + e)*M]); 
        //#pragma unroll

        for (unsigned int i = 0; i < SUB_N; i++ ){
                out[i] += temp * B_shared[i][e];
        }
      //}
    }

    __syncthreads();
  }
   #pragma unroll
  for(unsigned int i = 0; i < SUB_N; i++ ){
        if(blockIdx.y*SUB_N+i<N)
            C[blockIdx.y*M*SUB_N + i*M + tidx] = out[i];
  }

}
__global__ void elementWise_fp16(int hiddenSize, float *tmp_h, float *tmp_i, fp16 *bias, fp16 *phole_i, fp16 *phole_f, fp16 *phole_o, float *h_out, float *i_out, float *c_in, float *c_out) {  
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   
   if (index >= hiddenSize) return;
   
   //int batch = index / hiddenSize;
   int gateIndex = (index % hiddenSize); //+ 4 * batch * hiddenSize;   
   
   float g[4];

   for (int i = 0; i < 4; i++) {
      g[i] = tmp_i[i * hiddenSize + gateIndex] + tmp_h[i * hiddenSize + gateIndex];
      g[i] +=  fp16_to_fp32_gpu(bias[i * hiddenSize + index % hiddenSize]);
   }  

    g[1] += c_in[index]* fp16_to_fp32_gpu(phole_i[index]);
    g[2] += c_in[index]* fp16_to_fp32_gpu(phole_f[index]);
    //g[3] += c_in[index]*phole_o[index];
        
   
   float in_gate2    = tanhf(g[0]);   
   float in_gate     = sigmoidf(g[1]);
   float forget_gate = sigmoidf(g[2]);

   float out_gate    = g[3];
   
   float val = (forget_gate * c_in[index]) + (in_gate * in_gate2);
     
   c_out[index] = val;
   out_gate += val* fp16_to_fp32_gpu(phole_o[index]);
   out_gate = sigmoidf(out_gate);
   val = out_gate * tanhf(val);                                   

   h_out[index] = val;
   i_out[index] = val;
}

// Pointwise functions
void rand_generate_w( fp16 *y,  int n) {
  dim3 Gr;
  dim3 Bl;
  Bl.x = 128;
  Gr.x = (n + Bl.x - 1) / Bl.x;
    
  rand_generate<<<Gr, Bl>>>(y, n);

}
void pw_biasAdd_w(dim3 Gr, dim3 Bl, hipStream_t stream, float *y, float *bias, int n, int nBias){
    pw_biasAdd<<<Gr, Bl, 0, stream>>>(y, bias, n, nBias);
  }

void pw_vecAdd_w(dim3 Gr, dim3 Bl, hipStream_t stream, float *y, float *a,  float *b, int n){
  pw_vecAdd<<<Gr, Bl, 0, stream>>>(y,a, b, n);
}

void pw_vecMul_w(dim3 Gr, dim3 Bl, hipStream_t stream, float *y, float *a,  float *b, int n){
  pw_vecMul<<<Gr, Bl, 0, stream>>>(y, a, b, n);
}

void pw_tanh_w(dim3 Gr, dim3 Bl, hipStream_t stream, float *y, float *a, int n){
  pw_tanh<<<Gr, Bl, 0, stream>>>(y, a, n); 
}

void pw_sigmoid_w(dim3 Gr, dim3 Bl, hipStream_t stream, float *y, float *a, int n){
  pw_sigmoid<<<Gr, Bl, 0, stream>>>(y, a, n);
}
void add_pw_vecMul_w(dim3 Gr, dim3 Bl, hipStream_t stream, float *y, float *a,  float *b, int n){
  add_pw_vecMul<<<Gr, Bl, 0, stream>>>(y, a, b, n); 
}

void initKernel_w(dim3 Gr, dim3 Bl, hipStream_t stream, float * devPtr, const int val, int nwords){
 initKernel<<<Gr, Bl, 0, stream>>>(devPtr, val, nwords);
}

void testfloat_w(dim3 Gr, dim3 Bl, hipStream_t stream, float * devPtr, float * devPtr1){

testfloat<<<Gr, Bl, 0, stream>>>(devPtr, devPtr1);
   
}

void elementWise_fp_w(dim3 Gr, dim3 Bl, hipStream_t stream,int hiddenSize, float *tmp_h, float *tmp_i, float *bias, float *phole_i, float *phole_f, float *phole_o, float *h_out, float *i_out, float *c_in, float *c_out){
  elementWise_fp<<<Gr,Bl,0,stream>>>(hiddenSize,tmp_h, tmp_i, bias, phole_i, phole_f, phole_o, h_out, i_out, c_in, c_out);
}

void elementWise_fp_1_w(dim3 Gr, dim3 Bl, hipStream_t stream, int hiddenSize,
                               float *tmp_h, 
                               float *tmp_i, 
                               float *bias,
                               float *phole_i,
                               float *phole_f,
                               float *c_in) {
  elementWise_fp_1<<<Gr,Bl,0,stream>>>(hiddenSize, tmp_h, tmp_i, bias, phole_i, phole_f, c_in);
}

void elementWise_fp_2_w(dim3 Gr, dim3 Bl, hipStream_t stream, int hiddenSize,
                               float *tmp_i, 
                               float *c_in,
                               float *phole_o,
                               float *h_out,
                               float *i_out,
                               float *c_out){
  elementWise_fp_2<<<Gr,Bl,0,stream>>>(hiddenSize, tmp_i, c_in, phole_o, h_out, i_out, c_out);
} 

void add_vec_to_rows_w(dim3 Gr, dim3 Bl, hipStream_t stream, float alpha, float* row, float beta, float* dst, int nrow, int ncol) {
  add_vec_to_rows<<<Gr,Bl,0,stream>>>(alpha, row, beta, dst, nrow, ncol);
}

void softmax_reduce_w(dim3 Gr, dim3 Bl, hipStream_t stream, float*y, float*x, int nrow, int ncol){
_softmax_reduce<<<Gr,Bl,0,stream>>>(y, x, nrow, ncol);
}

void mygemv16( hipStream_t stream, const fp16*   dA, const float*  dx,float*  dy, const uint_t nRows, const uint_t nx){
 dim3 dim_grid((nRows + BLOCK_SIZE - 1) / BLOCK_SIZE);
  dim3 dim_block(BLOCK_SIZE);
  matvec_16<<<dim_grid, dim_block,0,stream>>>(dA, dx, dy, nRows, nx);

}
void mygemm16(hipStream_t stream, const fp16 *A, const float *B, float *C, int M, int N, int K){

  dim3 threadsPerBlock(SUB_M);
  dim3 blocksPerGrid((M + SUB_M - 1) / SUB_M, (N + SUB_N - 1) / SUB_N);
    
  matmat_16<<<blocksPerGrid,threadsPerBlock,0,stream>>>(A, B, C, M, K, N);
}
void elementWise_fp16_w(dim3 Gr, dim3 Bl, hipStream_t stream,int hiddenSize, float *tmp_h, float *tmp_i, fp16 *bias, fp16 *phole_i, fp16 *phole_f, fp16 *phole_o, float *h_out, float *i_out, float *c_in, float *c_out){
  elementWise_fp16<<<Gr,Bl,0,stream>>>(hiddenSize,tmp_h, tmp_i, bias, phole_i, phole_f, phole_o, h_out, i_out, c_in, c_out);
}

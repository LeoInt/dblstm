#include "layer.h"
#define impl 2
// 1 basic, 2 fused
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
static void cudaErrCheck_(hipError_t stat, const char *file, int line) {
  if (stat != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
  }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
static void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
  }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
static void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
  if (stat != HIPRAND_STATUS_SUCCESS) {
    fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
  }
}


 Layer::Layer(int input_dim, int output_dim, int cell_dim) { 
      input_dim_= input_dim; // 640
      output_dim_= output_dim; //640
      cell_dim_ = cell_dim; //320
        //cuda malloc
  
  cudaErrCheck(hipMalloc((void**)&wei_gifo_x_fw_,  input_dim_  * 4 * cell_dim_ * sizeof(fp16)));
  cudaErrCheck(hipMalloc((void**)&wei_gifo_m_fw_,  cell_dim_  * 4 * cell_dim_ * sizeof(fp16)));
  cudaErrCheck(hipMalloc((void**)&bias_fw_, 4 * cell_dim_ * sizeof(fp16)));
  cudaErrCheck(hipMalloc((void**)&phole_o_c_fw_, cell_dim_ * sizeof(fp16)));
  cudaErrCheck(hipMalloc((void**)&phole_i_c_fw_, cell_dim_ * sizeof(fp16)));
  cudaErrCheck(hipMalloc((void**)&phole_f_c_fw_, cell_dim_ * sizeof(fp16)));

  cudaErrCheck(hipMalloc((void**)&wei_gifo_x_bw_,  input_dim_  * 4 * cell_dim_ * sizeof(fp16)));
  cudaErrCheck(hipMalloc((void**)&wei_gifo_m_bw_,  cell_dim_  * 4 * cell_dim_ * sizeof(fp16)));
  cudaErrCheck(hipMalloc((void**)&bias_bw_, 4 * cell_dim_ * sizeof(fp16)));
  cudaErrCheck(hipMalloc((void**)&phole_o_c_bw_, cell_dim_ * sizeof(fp16)));
  cudaErrCheck(hipMalloc((void**)&phole_i_c_bw_,cell_dim_ * sizeof(fp16)));
  cudaErrCheck(hipMalloc((void**)&phole_f_c_bw_, cell_dim_ * sizeof(fp16)));
    }
  Layer::~Layer(){
  
  cudaErrCheck(hipFree(wei_gifo_x_fw_));
  cudaErrCheck(hipFree(wei_gifo_m_fw_));
  cudaErrCheck(hipFree(wei_gifo_x_bw_));
  cudaErrCheck(hipFree(wei_gifo_m_bw_));
  cudaErrCheck(hipFree(bias_fw_));
  cudaErrCheck(hipFree(bias_bw_));
  cudaErrCheck(hipFree(phole_o_c_fw_));
  cudaErrCheck(hipFree(phole_i_c_fw_));
  cudaErrCheck(hipFree(phole_f_c_fw_));
  cudaErrCheck(hipFree(phole_o_c_bw_));
  cudaErrCheck(hipFree(phole_i_c_bw_));
  cudaErrCheck(hipFree(phole_f_c_bw_));
  
  } 


#if impl==2
  float Layer::Propagate(hipblasHandle_t handle, float* in, float* out, int seqLength, float* tmp_h_fw, float* tmp_i_fw, float* tmp_h_bw, float* tmp_i_bw, float* h_data, float* c_data, float* h_data_bw, float* c_data_bw){
      //continue
  dim3 blockDim, gridDim;
  hipStream_t stream=NULL;
  int frame;
  hipEvent_t start, stop;
  float elapsedTime=0.f;
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  const hipblasOperation_t transa = HIPBLAS_OP_N;
  const hipblasOperation_t transb = HIPBLAS_OP_N;
  hipMemset(h_data, 0, cell_dim_*sizeof(float));
  hipMemset(c_data, 0, cell_dim_*sizeof(float));
  hipMemset(h_data_bw , 0, cell_dim_*sizeof(float));
  hipMemset(c_data_bw , 0, cell_dim_*sizeof(float));

  
  mygemm16(stream, wei_gifo_x_fw_, in, tmp_i_fw, 4*cell_dim_, seqLength, input_dim_);

  for (frame=0; frame < seqLength ; frame ++){
         mygemv16( stream, wei_gifo_m_fw_, h_data + (frame%2) * cell_dim_,tmp_h_fw, 4*cell_dim_, cell_dim_);
    
    stream=NULL;
    blockDim.x = 128;
    gridDim.x = (cell_dim_ + blockDim.x - 1) / blockDim.x;
    
    elementWise_fp16_w( gridDim, blockDim, stream, cell_dim_,tmp_h_fw, 
                       tmp_i_fw + 4 * frame * cell_dim_, 
                       bias_fw_,
                       phole_i_c_fw_,
                       phole_f_c_fw_,
                       phole_o_c_fw_,
                       h_data + ((frame + 1)%2) * cell_dim_,
                       out + frame * 2* cell_dim_,
                       c_data + (frame%2) * cell_dim_,
                       c_data + ((frame + 1)%2) * cell_dim_);
    }//frame loop

////backward//////////////////////////////////////////////////////////////////////
  mygemm16(stream, wei_gifo_x_bw_, in, tmp_i_bw, 4*cell_dim_, seqLength, input_dim_);

  /*cublasErrCheck(hipblasSgemm(handle,
                        transa, transb,
                        4 * cell_dim_, //m, number of rows of matrix op(A) and C.
                        seqLength , //n, number of cols of matrix op(B) and C.
                        input_dim_,  //k, number of cols of matrix op(A) and rows of op(B).
                        &alpha,
                        wei_gifo_x_bw_,
                        transa == HIPBLAS_OP_N ? 4*cell_dim_ : input_dim_,  //leading dimension = number of rows (I use the number of col because I do the transpose with transa)
                        in,
                        input_dim_,
                        &beta,
                        tmp_i_bw,
                        4 * cell_dim_));*/
  for (frame=seqLength-1; frame >=0 ; frame --){
         mygemv16( stream, wei_gifo_m_bw_, h_data_bw + ((frame+1)%2) * cell_dim_,tmp_h_bw, 4*cell_dim_, cell_dim_);
         /*cublasErrCheck(hipblasSgemv(handle, transa,
               4*cell_dim_, //m, number of rows of matrix op(A) 
               cell_dim_, //n, number of cols of matrix op(A).
               &alpha,
               wei_gifo_m_bw_, 
               4*cell_dim_,
               h_data_bw + ((frame+1)%2) * cell_dim_, 
               1, //stride input array
               &beta,
               tmp_h_bw, 
               1)); //stide output array*/
    hipStream_t stream=NULL;
    blockDim.x = 128;
    gridDim.x = (cell_dim_ + blockDim.x - 1) / blockDim.x;
    
    elementWise_fp16_w( gridDim, blockDim, stream, cell_dim_,tmp_h_bw, 
                       tmp_i_bw + 4 * frame * cell_dim_, 
                       bias_bw_,
                       phole_i_c_bw_,
                       phole_f_c_bw_,
                       phole_o_c_bw_,
                       h_data_bw + (frame%2) * cell_dim_,
                       out + frame * 2* cell_dim_ + cell_dim_,
                       c_data_bw + ((frame+1)%2) * cell_dim_,
                       c_data_bw + (frame%2) * cell_dim_);
  }
/////////////////////////////frame loop
hipEventRecord(stop);
hipEventSynchronize(stop);
hipEventElapsedTime(&elapsedTime, start, stop);
return elapsedTime;  
}
#endif  

#if impl==4
  float Layer::Propagate(hipblasHandle_t handle, float* in, float* out, int seqLength, float* tmp_h_fw, float* tmp_i_fw, float* tmp_h_bw, float* tmp_i_bw, float* h_data, float* c_data, float* h_data_bw, float* c_data_bw){
      //continue
  dim3 blockDim, gridDim;
  hipStream_t stream_fw, stream_bw;
  hipStreamCreate(&stream_fw);
  hipStreamCreate(&stream_bw);
  int frame, frame_bw;
  float alpha = 1.f;
  float beta  = 0.f;
  hipEvent_t start, stop;
  float elapsedTime=0.f;
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  const hipblasOperation_t transa = HIPBLAS_OP_N;
  const hipblasOperation_t transb = HIPBLAS_OP_N;

  hipMemset(h_data, 0, cell_dim_*sizeof(float));
  hipMemset(c_data, 0, cell_dim_*sizeof(float));
  hipMemset(h_data_bw , 0, cell_dim_*sizeof(float));
  hipMemset(c_data_bw , 0, cell_dim_*sizeof(float));

  
   hipblasSetStream(handle, stream_fw);
 
  cublasErrCheck(hipblasSgemm(handle,
                        transa, transb,
                        4 * cell_dim_, //m, number of rows of matrix op(A) and C.
                        seqLength , //n, number of cols of matrix op(B) and C.
                        input_dim_,  //k, number of cols of matrix op(A) and rows of op(B).
                        &alpha,
                        wei_gifo_x_fw_,
                        transa == HIPBLAS_OP_N ? 4 * cell_dim_ : input_dim_,  //leading dimension = number of rows (I use the number of col because I do the transpose with transa)
                        in,
                        input_dim_,
                        &beta,
                        tmp_i_fw,
                        4 * cell_dim_));
  hipblasSetStream(handle, stream_bw);
 
   cublasErrCheck(hipblasSgemm(handle,
                        transa, transb,
                        4 * cell_dim_, //m, number of rows of matrix op(A) and C.
                        seqLength , //n, number of cols of matrix op(B) and C.
                        input_dim_,  //k, number of cols of matrix op(A) and rows of op(B).
                        &alpha,
                        wei_gifo_x_bw_,
                        transa == HIPBLAS_OP_N ? 4*cell_dim_ : input_dim_,  //leading dimension = number of rows (I use the number of col because I do the transpose with transa)
                        in,
                        input_dim_,
                        &beta,
                        tmp_i_bw,
                        4 * cell_dim_));
 for (frame=0, frame_bw=seqLength-1; frame < seqLength ; frame ++, frame_bw --){
         hipblasSetStream(handle, stream_fw);
   
         cublasErrCheck(hipblasSgemv(handle, transa,
               4 * cell_dim_, //m, number of rows of matrix op(A) 
               cell_dim_, //n, number of cols of matrix op(A).
               &alpha,
               wei_gifo_m_fw_, 
               4 * cell_dim_,
               h_data + (frame%2) * cell_dim_, 
               1, //stride input array
               &beta,
               tmp_h_fw, 
               1)); //stide output array
    
    
    hipblasSetStream(handle, stream_bw);
           cublasErrCheck(hipblasSgemv(handle, transa,
               4*cell_dim_, //m, number of rows of matrix op(A) 
               cell_dim_, //n, number of cols of matrix op(A).
               &alpha,
               wei_gifo_m_bw_, 
               4*cell_dim_,
               h_data_bw + ((frame_bw+1)%2) * cell_dim_, 
               1, //stride input array
               &beta,
               tmp_h_bw, 
               1)); //stide output array
    
   
    blockDim.x = 128;
    gridDim.x = (cell_dim_ + blockDim.x - 1) / blockDim.x;
    
    elementWise_fp_w( gridDim, blockDim, stream_fw, cell_dim_,tmp_h_fw, 
                       tmp_i_fw + 4 * frame * cell_dim_, 
                       bias_fw_,
                       phole_i_c_fw_,
                       phole_f_c_fw_,
                       phole_o_c_fw_,
                       h_data + ((frame + 1)%2) * cell_dim_,
                       out + frame * 2* cell_dim_,
                       c_data + (frame%2) * cell_dim_,
                       c_data + ((frame + 1)%2) * cell_dim_);
       elementWise_fp_w( gridDim, blockDim, stream_bw, cell_dim_,tmp_h_bw, 
                       tmp_i_bw + 4 * frame_bw * cell_dim_, 
                       bias_bw_,
                       phole_i_c_bw_,
                       phole_f_c_bw_,
                       phole_o_c_bw_,
                       h_data_bw + (frame_bw%2) * cell_dim_,
                       out + frame_bw * 2* cell_dim_ + cell_dim_,
                       c_data_bw + ((frame_bw+1)%2) * cell_dim_,
                       c_data_bw + (frame_bw%2) * cell_dim_);
 
    }//frame loop

/////////////////////////////frame loop
hipEventRecord(stop);
hipEventSynchronize(stop);
hipEventElapsedTime(&elapsedTime, start, stop);
return elapsedTime;  
}
#endif  


  void Layer::Init() {

  rand_generate_w(wei_gifo_x_fw_, input_dim_  * 4 * cell_dim_);
  rand_generate_w(wei_gifo_m_fw_,  cell_dim_  * 4 * cell_dim_);
  rand_generate_w(bias_fw_, 4 * cell_dim_);
  rand_generate_w(phole_o_c_fw_, cell_dim_);
  rand_generate_w(phole_f_c_fw_, cell_dim_);
  rand_generate_w(phole_i_c_fw_, cell_dim_);
  rand_generate_w(wei_gifo_x_bw_, input_dim_  * 4 * cell_dim_);
  rand_generate_w(wei_gifo_m_bw_,  cell_dim_  * 4 * cell_dim_);
  rand_generate_w(bias_bw_, 4 * cell_dim_);
  rand_generate_w(phole_o_c_bw_, cell_dim_);
  rand_generate_w(phole_f_c_bw_, cell_dim_);
  rand_generate_w(phole_i_c_bw_, cell_dim_);
  }


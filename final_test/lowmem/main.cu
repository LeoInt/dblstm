#include "layer.h"
#include "AffineTransform.h"
#include "net.h"
#include <stdlib.h>
#include <stdio.h>
#define DEBUG 0

static void Print_matrix_to_file(const char nmfile[], float* mat, int n, int m, int r_c){
    //const char nmfile[] = "out.txt";
    std::ofstream outseis(nmfile); // output, normal file
    float *data_host;
    data_host=(float*)malloc(n*m*sizeof(float));
    hipMemcpy(data_host, mat, n*m*sizeof(float), hipMemcpyDeviceToHost);  // this won't work, will throw error
    if(r_c==0){
      for (int jj=0; jj<n; jj++)
      {
        std::stringstream buf;
        int ii;
        for (ii=0; ii<m; ii++)
        {
          float* temp=(float *)(data_host+jj*m+ii);
                    //printf("%f ", temp);
          buf<< *temp <<" ";
                  //if(jj==101) printf("%f ", *temp);
        }
        outseis << buf.str() << "\n";
            //printf("\n%d %d row, col", jj, ii);
      }
    }else{
      for (int jj=0; jj<n; jj++)
      {
        std::stringstream buf;
        int ii;
        for (ii=0; ii<m; ii++)
        {
          float* temp=(float *)(data_host+ii*n+jj);
                    //printf("%f ", temp);
          buf<< *temp <<" ";
                  //if(jj==101) printf("%f ", *temp);
        }
        outseis << buf.str() << "\n";
            //printf("\n%d %d row, col", jj, ii);
      }
    }
    free(data_host);
  }


static  void Print_matrix(float* mat, int n, int m, int r_c){
    //const char nmfile[] = "out.txt";
    float *data_host;
    data_host=(float*)malloc(n*m*sizeof(float));
    hipMemcpy(data_host, mat, n*m*sizeof(float), hipMemcpyDeviceToHost);  // this won't work, will throw error
    if(r_c==0){
      for (int jj=0; jj<n; jj++)
      {
        int ii;
        for (ii=0; ii<m; ii++)
        {
          float* temp=(float *)(data_host+jj*m+ii);
          printf("%.10e ", *temp);
                  //if(jj==101) printf("%f ", *temp);
        }
        printf("\n");
      }
    }else{
      for (int jj=0; jj<n; jj++)
      {
        int ii;
        for (ii=0; ii<m; ii++)
        {
          float* temp=(float *)(data_host+ii*n+jj);
          printf("%.10e ", *temp);
                    //if(jj==101) printf("%f ", *temp);
        }
        printf("\n");
      }
    }
    free(data_host);
  }


int main(int argc, char* argv[]) {
   int seqLength=100;
   int numLayers=4;
   int hiddenSize=320;
   int input_dim=2*hiddenSize;
   int output_dim=46;
   float* x_in;
   float* x_out;
   float* x_in_d;
   float* x_out_d;
   float* x_out_soft;

   if (argc == 4) {
      seqLength = atoi(argv[1]);
      numLayers =  atoi(argv[2]);
      hiddenSize =  atoi(argv[3]);
   }
   else if (argc == 1) {
      seqLength = 100;
      numLayers = 4;
      hiddenSize = 320;
   }
   
   //x_in = (float*)malloc(seqLength*input_dim*sizeof(float));
   hipHostMalloc((void**)&x_in, seqLength*input_dim*sizeof(float)); //pinned memory
   hipHostMalloc((void**)&x_out, seqLength*output_dim*sizeof(float)); //pinned memory
   //x_out = (float*)malloc(seqLength*output_dim*sizeof(float));  
   
   hipMalloc((void**)&x_in_d, seqLength * input_dim * sizeof(float));
   hipMalloc((void**)&x_out_d, seqLength * output_dim * sizeof(float));
   hipMalloc((void**)&x_out_soft, seqLength * output_dim * sizeof(float));
   
   srand (time(NULL));
  	
   for(int i=0; i<seqLength * input_dim; i++){
   		x_in[i]=(rand() % 10)/10.f;
   		//printf("%f ", x_in[i]);
   }
   
   Net* N;
   Layer* L;
   AffineTransform* A;
   N = new Net(4,hiddenSize);
   N->Resize(seqLength); 

   	for(int i=0; i<numLayers; i++){
   		if(i==0)
			L = new Layer(input_dim,2*hiddenSize,hiddenSize);
		else
			L = new Layer(2*hiddenSize,2*hiddenSize,hiddenSize);
		L->Init();
		if(DEBUG){	
   			char s1[2]="H";
   			char s2[5]=".txt";
   			char s3[5]="bias";
   			char s4[3]="Wx";
   			char s5[3]="Wh";
   			char pi[8]="phole_i";
   			char pf[8]="phole_f";
   			char po[8]="phole_o";
   			char result[19];
   			char result1[13];
   			char result2[11];
   			sprintf(result1,"%s%d%s",s3,i,s2);
   			Print_matrix_to_file(result1, L->bias_fw(), 4*hiddenSize, 1, 1);
   			sprintf(result2,"%s%d%s",s4,i,s2);
   			if(i==0)
   				Print_matrix_to_file(result2, L->wei_gifo_x_fw(), 4*hiddenSize, input_dim, 1);
   			else
   				Print_matrix_to_file(result2, L->wei_gifo_x_fw(), 4*hiddenSize, 2*hiddenSize, 1);
   			sprintf(result2,"%s%d%s",s5,i,s2);
   			Print_matrix_to_file(result2, L->wei_gifo_m_fw(), 4*hiddenSize, hiddenSize, 1); 
 			sprintf(result,"%s%d%s",pi,i,s2);
   			Print_matrix_to_file(result, L->phole_i_c_fw(), hiddenSize, 1, 1);
   			sprintf(result,"%s%d%s",pf,i,s2);
   			Print_matrix_to_file(result, L->phole_f_c_fw(), hiddenSize, 1, 1);
   			sprintf(result,"%s%d%s",po,i,s2);
   			Print_matrix_to_file(result, L->phole_o_c_fw(), hiddenSize, 1, 1);
  	
  			char s6[8]="bw.txt";
   			sprintf(result1,"%s%d%s",s3,i,s6);
   			Print_matrix_to_file(result1, L->bias_bw(), 4*hiddenSize, 1, 1);
   			sprintf(result2,"%s%d%s",s4,i,s6);
   			if(i==0)
   				Print_matrix_to_file(result2, L->wei_gifo_x_bw(), 4*hiddenSize, input_dim, 1);
   			else
   				Print_matrix_to_file(result2, L->wei_gifo_x_bw(), 4*hiddenSize, 2*hiddenSize, 1);
   			sprintf(result2,"%s%d%s",s5,i,s6);
   			Print_matrix_to_file(result2, L->wei_gifo_m_bw(), 4*hiddenSize, hiddenSize, 1); 
 			sprintf(result,"%s%d%s",pi,i,s6);
   			Print_matrix_to_file(result, L->phole_i_c_bw(), hiddenSize, 1, 1);
   			sprintf(result,"%s%d%s",pf,i,s6);
   			Print_matrix_to_file(result, L->phole_f_c_bw(), hiddenSize, 1, 1);
   			sprintf(result,"%s%d%s",po,i,s6);
   			Print_matrix_to_file(result, L->phole_o_c_bw(), hiddenSize, 1, 1);		
 		}
		N->AppendLayer(L);
   	}
   	A = new AffineTransform(2*hiddenSize, output_dim); 
   	A->Init();
   	if(DEBUG){
		Print_matrix_to_file("WA.txt", A->wei_affine(), output_dim, input_dim, 1);
   		Print_matrix_to_file("bA.txt", A->bias(), output_dim, 1, 1);
	} 	
	N->AppendAffineTransformLayer(A);
 	hipMemcpy( x_in_d, x_in, seqLength * input_dim *sizeof(float), hipMemcpyHostToDevice);
 	if(DEBUG){Print_matrix_to_file("X.txt", x_in_d, input_dim, seqLength, 1);}
 	hipEvent_t start, stop;
	float elapsedTime=0.f;
	hipblasHandle_t handle;
	hipStream_t stream=NULL;
  	size_t dimBlock = output_dim > CU1DBLOCK ? CU1DBLOCK : output_dim;
    size_t dimGrid = seqLength;
    

  	hipblasCreate(&handle);
    

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
 	hipMemcpy( x_in_d, x_in, seqLength * input_dim *sizeof(float), hipMemcpyHostToDevice);
 	N->Feedforward(handle, x_in_d, x_out_d, seqLength);
 	softmax_reduce_w(dimGrid, dimBlock, stream, x_out_soft, x_out_d, seqLength, output_dim);
 	hipMemcpy( x_out, x_out_soft, seqLength * output_dim *sizeof(float), hipMemcpyDeviceToHost);
 	hipEventRecord(stop);
 	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("%f ", elapsedTime);
 	if(DEBUG){
	Print_matrix_to_file("X1.txt", N->PropagateBuffer(), 2 * hiddenSize, seqLength, 1);
 	Print_matrix_to_file("X2.txt", N->PropagateBuffer() + 2 * hiddenSize*seqLength, 2 * hiddenSize, seqLength, 1);
 	Print_matrix_to_file("Xout.txt", x_out_d, output_dim, seqLength, 1);
 	Print_matrix_to_file("Xsoft.txt", x_out_soft, output_dim, seqLength, 1);
	} 	
 	delete N;
 	//cudaErrCheck(hipMemcpy( devciao1, ciao1, sizeof(float), hipMemcpyHostToDevice));    
}
